#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

#define NUM_FEATURES 1766
#define NUM_CHEMICALS 3000

// Function to read data from a text file into a 2D array
void readDataFromFile(const std::string& filename, float* data, int rows, int cols) {
    std::ifstream file(filename);
    if (file.is_open()) {
        std::string line;
        int row = 0;
        while (std::getline(file, line) && row < rows) {
            std::istringstream iss(line);
            for (int col = 0; col < cols; ++col) {
                iss >> data[row * cols + col];
            }
            row++;
        }
        file.close();
    } else {
        std::cerr << "Unable to open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }
}

__global__ void computeDistanceAndSimilarity(float *bioresponse_matrix, float *known_drug, float *D, float *S) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < NUM_CHEMICALS) {
        float distance = 0.0f;
        for (int i = 0; i < NUM_FEATURES; ++i) {
            distance += pow(fabs(bioresponse_matrix[idx * NUM_FEATURES + i] - known_drug[i]), 1.50);
        }
        D[idx] = pow(distance, 1.0f / 1.50f);

        // Compute similarity
        float gamma = 1.0f / NUM_FEATURES;
        S[idx] = expf(-D[idx] * gamma);
    }
}

int main() {
    // Allocate memory for data
    float *bioresponse_matrix, *known_drug, *D, *S;
    hipMallocManaged(&bioresponse_matrix, NUM_CHEMICALS * NUM_FEATURES * sizeof(float));
    hipMallocManaged(&known_drug, NUM_FEATURES * sizeof(float));
    hipMallocManaged(&D, NUM_CHEMICALS * sizeof(float));
    hipMallocManaged(&S, NUM_CHEMICALS * sizeof(float));

    // Read data from files and populate bioresponse_matrix and known_drug arrays
    readDataFromFile("bioresponse_descriptors_matrix.txt", bioresponse_matrix, NUM_CHEMICALS, NUM_FEATURES);
    readDataFromFile("known_drug.txt", known_drug, 1, NUM_FEATURES);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (NUM_CHEMICALS + blockSize - 1) / blockSize;
    computeDistanceAndSimilarity<<<numBlocks, blockSize>>>(bioresponse_matrix, known_drug, D, S);
    hipDeviceSynchronize();

    // Allocate memory for sorted indices
    int *sorted_indices;
    hipMallocManaged(&sorted_indices, NUM_CHEMICALS * sizeof(int));

    // Use Thrust to sort the similarity vector S along with their corresponding indices
    thrust::device_ptr<float> dev_ptr_S(S);
    thrust::sequence(sorted_indices, sorted_indices + NUM_CHEMICALS);
    thrust::sort_by_key(dev_ptr_S, dev_ptr_S + NUM_CHEMICALS, sorted_indices, thrust::greater<float>());

    // Output top 10 most similar chemicals
    std::cout << "Top 10 most similar chemicals:\n";
    for (int i = 0; i < 10; ++i) {
        std::cout << sorted_indices[i] << "\n";
    }

    // Free allocated memory
    hipFree(bioresponse_matrix);
    hipFree(known_drug);
    hipFree(D);
    hipFree(S);
    hipFree(sorted_indices);

    return 0;
}

